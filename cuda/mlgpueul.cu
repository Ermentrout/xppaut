/* nvcc -lcublas -o MLGPU.SO --shared -Xcompiler -fpic mlgpu.cu 
  export PATH=/usr/local/cuda-7.0/bin:$PATH
 export LD_LIBRARY_PATH=/usr/local/cuda-7.0/lib64:$LD_LIBRARY_PATH

This one uses euler to iterate over a ceratin number of points
the dt and the nsteps is included in the parameters

*/

#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h> 
#include <getopt.h>
#include <unistd.h>
#include <sys/time.h>

#include "hipblas.h"
#define MAXPAR 50
#define NTHREAD 128

#define real float

__constant__ real devp[MAXPAR];



real *devsum,*devy,*devyp,*devwgt;
/* needed for double to float */
real *hosty,*hostyp,*hostwgt;

int allocflag=0;
int recopywgtflag=0;

hipblasHandle_t handle;
/* ML parameters   */ 

#define iapp devp[0]
#define phi devp[1]
#define va devp[2]
#define vb devp[3]
#define vc devp[4]
#define vd devp[5]
#define gca devp[6]
#define vk devp[7]
#define vl devp[8]
#define gk devp[9]
#define gl devp[10]
#define tsyn devp[11]
#define gsyn devp[12]
#define vsyn devp[13]
#define vth devp[14]
#define vshp devp[15]
#define deltat devp[16]
/*  ML Functions */

__device__ real minf(real v)
{
  return .5*(1+tanhf((v-va)/vb));
}
__device__ real ninf(real v)
{

  return .5*(1+tanhf((v-vc)/vd));
}
__device__ real lamn(real  v)
{
  return phi*coshf((v-vc)/(2*vd));
}
__device__ real s_inf(real v)
{
  
  return 1.0/(1.0+expf(-(v-vth)/vshp)); 

}

/*  this is the right-hand side evaluation */



__global__ void update_rhs(real *y,real *yp,real *stot, int n)
{
    int i=blockIdx.x*blockDim.x+threadIdx.x;
    real v,w,s;
 /*   if(i<10){
    printf(" %d %g %g %g \n",i,y[i],y[i+n],y[i+2*n]);
    }
  */
    if(i<n){
     v=y[i];
     w=y[i+n];
     s=y[i+2*n];
     yp[i]=iapp-gl*(v-vl)-gca*(v-1.0)*minf(v)-gk*w*(v-vk)-gsyn*stot[i]*(v-vsyn);
     yp[i+n]=lamn(v)*(ninf(v)-w);
     yp[i+2*n]=(sinf(v)-s)/tsyn;
     y[i]+=deltat*yp[i];
     y[i+n]+=deltat*yp[i+n];
     y[i+2*n]+=deltat*yp[i+2*n];

     
    }
}


/*  I will assume for now that the weight do not change throughout the simulation
    so they will only be loaded into the device once
*/
void allocate_ram(int n, double *w)
{
  int n3=3*n,nn=n*n;
  int i;
  if(allocflag==1)return;  /* already allocated */
  hostwgt=(real *)malloc(nn*sizeof(float));
  hipMalloc((void**)&devwgt,nn*sizeof(real));
  /*  copy one time only for now */
  for(i=0;i<nn;i++)
      hostwgt[i]=(real)w[i];
  hipMemcpy(devwgt,hostwgt,nn*sizeof(real),hipMemcpyHostToDevice); 
  hosty=(real *)malloc(n3*sizeof(real));
  hipMalloc((void**)&devy,n3*sizeof(real));
  hostyp=(real *)malloc(n3*sizeof(real));
  hipMalloc((void**)&devyp,n3*sizeof(real));
    
  hipMalloc((void**)&devsum,n*sizeof(real));
 hipblasCreate(&handle);
  allocflag=1;  

}

/* do this every time */
void copy_par_to_dev(double *par)
{
  real dp[MAXPAR];
  int i;
  for(i=0;i<MAXPAR;i++)
    dp[i]=(real)par[i];
  hipMemcpyToSymbol(HIP_SYMBOL(devp),dp,sizeof(dp));
}

/* do this every time */

void copy_to_dev(int n,double  *w,double *y)
{
  int i;
  int n3=3*n,nn=n*n;

  for(i=0;i<n3;i++)
    hosty[i]=(float)y[i];
  hipMemcpy(devy,hosty,n3*sizeof(real),hipMemcpyHostToDevice);
  if(recopywgtflag){

    for(i=0;i<nn;i++)
      hostwgt[i]=(float)w[i];
  hipMemcpy(devwgt,hostwgt,nn*sizeof(real),hipMemcpyHostToDevice);
  }
}  

void copy_from_dev(double *ydot, int nn)
{
  int i;
  hipMemcpy(hostyp,devy,nn*sizeof(real),hipMemcpyDeviceToHost);
  for(i=0;i<nn;i++)
    ydot[i]=(double)hostyp[i];
}






void update_sums(int n)
{
  real alpha=1.0,beta=0.0;


  /* single  */
  hipblasSgemv(handle,HIPBLAS_OP_T,n,n,&alpha,devwgt,n,&devy[2*n],1,&beta,devsum,1);
  /* double */
  /* cublasDgemv(handle,CUBLAS_OP_T,n,n,&alpha,wgt,n,s,1,&beta,sum,1); */
 
}


extern "C"

void MLGPUEUL(int nn,int ivar, double *par,double *var,double *z[50],double *ydot)
{
  int n=nn/3;
  int niter=(int)par[17];
  int i;
  double *y;
  int nblock=(int)((n+NTHREAD-1)/NTHREAD);

  y=var+ivar;
  allocate_ram( n, z[0]);
  copy_par_to_dev(par);
  copy_to_dev(n,z[0],y);
  for(i=0;i<niter;i++){
  update_sums(n);
  update_rhs<<<nblock,128>>>(devy,devyp,devsum,n);
  }
  copy_from_dev(ydot,nn);
 
}


